#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

// Updated resolution to 1024×1024
constexpr int WIDTH     = 1024;
constexpr int HEIGHT    = 1024;
constexpr int N_SPHERES = 10;

// Simple 3-float vector
struct Vec {
    float x, y, z;
    __host__ __device__ Vec() : x(0), y(0), z(0) {}
    __host__ __device__ Vec(float X, float Y, float Z) : x(X), y(Y), z(Z) {}

    __host__ __device__ Vec operator+(Vec const &o) const { return {x+o.x, y+o.y, z+o.z}; }
    __host__ __device__ Vec operator-(Vec const &o) const { return {x-o.x, y-o.y, z-o.z}; }
    __host__ __device__ Vec operator*(float k)      const { return {x*k,   y*k,   z*k  }; }
    __host__ __device__ Vec operator-()             const { return Vec{-x, -y, -z}; }
};
__host__ __device__ inline Vec operator*(float k, Vec const &v) { return v * k; }

__host__ __device__ inline float dot(Vec const &a, Vec const &b) {
    return a.x*b.x + a.y*b.y + a.z*b.z;
}
__host__ __device__ inline Vec norm(Vec const &v) {
    float invLen = rsqrtf(dot(v,v));
    return {v.x * invLen, v.y * invLen, v.z * invLen};
}

// Sphere definition
struct Sphere {
    Vec   center;
    float radius;
    Vec   color;
};

// ray–sphere intersection
__device__ bool hit_sphere(Vec const &orig, Vec const &dir,
                           Sphere const &s, float &t) {
    Vec oc = orig - s.center;
    float b = dot(oc, dir);
    float c = dot(oc, oc) - s.radius * s.radius;
    float disc = b*b - c;
    if (disc < 0.0f) return false;
    t = -b - sqrtf(disc);
    return t > 0.0f;
}

// Phong shading
__host__ __device__ Vec phong_shade(
    Vec const &P, Vec const &N, Vec const &V,
    Vec const &lightPos, Vec const &Kd, Vec const &Ks, float shininess) {
    Vec ambient = 0.1f * Kd;
    Vec L       = norm(lightPos - P);
    float diff  = fmaxf(dot(N,L), 0.0f);
    Vec diffuse = diff * Kd;
    Vec R       = norm(2.0f * dot(N,L) * N - L);
    float spec  = powf(fmaxf(dot(R,V), 0.0f), shininess);
    Vec specular= spec * Ks;
    return ambient + diffuse + specular;
}

// render kernel: one thread per pixel
__global__ void render_kernel(Vec *fb, Sphere const *spheres) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= WIDTH || y >= HEIGHT) return;

    Vec origin   {0.0f,  0.0f,  2.0f};
    Vec lightPos{-5.0f, -5.0f, 10.0f};

    float u = -1.0f + 2.0f * x / (WIDTH  - 1);
    float v = -1.0f + 2.0f * y / (HEIGHT - 1);
    Vec dir = norm(Vec{u,v,0.0f} - origin);

    float best_t = 1e20f;
    int   hit_id = -1;
    #pragma unroll
    for(int i = 0; i < N_SPHERES; ++i) {
        float t;
        if (hit_sphere(origin, dir, spheres[i], t) && t < best_t) {
            best_t = t;
            hit_id = i;
        }
    }

    Vec pixelColor;
    if (hit_id >= 0) {
        Vec P  = origin + dir * best_t;
        Vec N  = norm(P - spheres[hit_id].center);
        Vec V  = norm(-dir);
        Vec Kd = spheres[hit_id].color;
        pixelColor = phong_shade(P, N, V, lightPos, Kd, {1,1,1}, 32.0f);
    } else {
        int ix = int(floor((u+1.0f)*5.0f));
        int iy = int(floor((v+1.0f)*5.0f));
        pixelColor = ((ix + iy)&1)==0 ? Vec{0.9f,0.9f,0.9f}
                                     : Vec{0.1f,0.1f,0.1f};
    }

    fb[y*WIDTH + x] = pixelColor;
}

inline int toByte(float c) {
    c = fminf(fmaxf(c,0.0f),1.0f);
    return int(c * 255.999f);
}

int main() {
    // prepare host spheres
    Sphere h_spheres[N_SPHERES];
    for(int i = 0; i < N_SPHERES; ++i) {
        float y = -1.0f + i * (2.0f/(N_SPHERES-1));
        float z = -2.0f - i * 0.5f;
        h_spheres[i].center = {0.0f, y, z};
        h_spheres[i].radius = 0.75f;
        float t = float(N_SPHERES - i) / N_SPHERES;
        h_spheres[i].color  = {t, 0.5f, 1.0f - t};
    }

    // allocate device spheres in global memory
    Sphere *d_spheres;
    hipMalloc(&d_spheres, N_SPHERES * sizeof(Sphere));
    hipMemcpy(d_spheres, h_spheres,
               N_SPHERES * sizeof(Sphere),
               hipMemcpyHostToDevice);

    // allocate device framebuffer
    Vec *d_fb;
    hipMalloc(&d_fb, WIDTH*HEIGHT * sizeof(Vec));

    // allocate pinned host memory
    Vec *h_fb;
    hipHostMalloc(&h_fb, WIDTH*HEIGHT * sizeof(Vec), hipHostMallocDefault);

    // launch kernel
    dim3 block(16,16);
    dim3 grid((WIDTH+block.x-1)/block.x,
              (HEIGHT+block.y-1)/block.y);
    render_kernel<<<grid,block>>>(d_fb, d_spheres);
    hipDeviceSynchronize();

    // copy framebuffer back to host
    hipMemcpy(h_fb, d_fb,
               WIDTH*HEIGHT * sizeof(Vec),
               hipMemcpyDeviceToHost);

    // output PPM
    std::cout << "P3\n" << WIDTH << ' ' << HEIGHT << "\n255\n";
    for(int y = HEIGHT-1; y >= 0; --y) {
        for(int x = 0; x < WIDTH; ++x) {
            Vec c = h_fb[y*WIDTH + x];
            std::cout
                << toByte(c.x) << ' '
                << toByte(c.y) << ' '
                << toByte(c.z) << '\n';
        }
    }

    // cleanup
    hipFree(d_spheres);
    hipFree(d_fb);
    hipHostFree(h_fb);
    return 0;
}
